#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstdlib>

using namespace std;


const int LOW = -50;
const int HIGH = 50;

float get_random_number(float min, float max) {
    return LOW + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (HIGH - LOW)));
}

/*
This kernel implements an online softmax operation on a matrix of size (M, N).
The softmax operation is performed on the last dimension of the matrix.

How this works:
In this, we handle each row with a block where the threads within one block work together
to process one row (max and norm factor). Each thread will process some elements
and will contains its local max and local norm in shared memory. Then, we perform reduction
operations to compute the final max and norm factor. Also, we compute maxes and norms
in one pass itself.
*/
__global__ void softmax_kernel(float* __restrict__ xd, float* __restrict__ resd, int M, int N) {
    // max and norm reduction will happen in shared memory (static)
    __shared__ float smem[1024];

    int row = blockIdx.x;
    int tid = threadIdx.x;

    printf("%d\n", blockIdx.x);

    // edge condition (we don't process further)
    if (row >= M) return;
    /**
     * Here, we are just getting the start index for this row that current block threads will be operating on
     */
    float* input_row = xd + row * N;
    float* output_row = resd + row * N;

    float local_max = -INFINITY;
    float local_norm = 0.0f;

    // compute local max and norm for each thread
    // and then finally have a sync barrier before moving on

    /**
     * Current block threads will work in parallel, each thread of them will calulate for elemnets reside at indices: {tid, tid + BLOCK_DIM,  tid + 2 * BLOCK_DIM}
     * 
     * This is helpful to achieve memory coalescing, where differnt threads operate on different elements, combine together for more efficient
     * execution for the operation they want with less memory transactions needed
     */
    for (int i = tid; i < N; i += blockDim.x) {
        float x = input_row[i];
        if (x > local_max) {
            local_norm *= expf(local_max - x);
            local_max = x;
        }
        local_norm += expf(x - local_max);
    }
    __syncthreads();

    // each thread will have its own local max
    // we store it in the tid of the shared memory

    /**
     * For each block (row) threads, each thread stores the maximum value it had among all values it operated on
     */
    smem[tid] = local_max;
    __syncthreads();

    // block-level reduction in O(log(N)) time over all threads
    // is faster than linear reduction over all threads
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            smem[tid] = max(smem[tid], smem[tid + stride]);
        }
        // sync barrier before next iteration to ensure correctness
        __syncthreads();
    }

    // the first element after max reduction from all threads
    // will contain the global max for the row
    float row_max = smem[0];
    __syncthreads();

    /**
     * This trick helps in calculating the norms in optimized fasion instead of 2 separate loops
     */
    smem[tid] = local_norm * expf(local_max - row_max);
    __syncthreads();

    // sum reduction similar to above for global norm factor
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }
    float row_norm = smem[0];
    __syncthreads();

    // finally, compute softmax
    for (int i = tid; i < N; i += blockDim.x) {
        output_row[i] = expf(input_row[i] - row_max) / row_norm;
       printf("%f ", output_row[i]); 
    }

    printf("\n");
}

/*
Runs the online softmax kernel: `id = 2`
*/
void start_kernel_execution(float* mat, int M, int N) {
    // grid size and block size for this kernel
    // change as necessary
    dim3 block_size(M);
    dim3 grid_size(M);

    float *matd, *resd;

    if (hipMalloc(&matd, M * N * sizeof(float)) != hipSuccess ||
        hipMalloc(&resd, M * N * sizeof(float)) != hipSuccess) {
        std::cerr << "Error: hipMalloc failed." << std::endl;
        return;
    }

    // Copy matrix from host to device
    if (hipMemcpy(matd, mat, M * N * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error: hipMemcpy failed." << std::endl;
        hipFree(matd);
        hipFree(resd);
        return;
    }

    softmax_kernel<<<grid_size, block_size>>>(matd, resd, M, N);

    hipFree(matd);
    hipFree(resd);
}

int main() {
    int M = 10;
    int N = 20;
    int matsize = M * N;
    int totalsize = matsize * sizeof(float);

    // allocate and initialize host matrix
    float* mat = (float*)malloc(totalsize);
    for (int i = 0; i < matsize; i++) {
        mat[i] = get_random_number(-10, 10);
    }

    start_kernel_execution(mat, M, N);

    free(mat);
}
